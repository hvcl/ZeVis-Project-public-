#include "hip/hip_runtime.h"
#include "calculator_subregion.cuh"


extern "C"
__device__ int _pow_(int a, int b) {
	int value = 1;
	for (int i = 0; i < b; ++i) {
		value *= a;
	}
	return value;
}

extern "C"
__global__ void _cuda_parallel_sum(unsigned char *in, int num_elements, int *sum, const float *information)
{
	//Holds intermediates in shared memory reduction
	__syncthreads();
	__shared__ int buffer[WARP_SIZE];
	int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int tile_size = (int)information[0] * (int)information[0];
	int level_size = _pow_(2, (int)information[11]);
	//information : block_width, tablesize, pos_x,y,z , up_x,y,z , global_x,y,z
	if ((globalIdx / tile_size) % level_size == 0) {

		float vx = -1 * information[2] + (information[8] + ((int)(globalIdx % tile_size) % (int)information[0]));
		float vy = -1 * information[3] + (information[9] + ((int)(globalIdx % tile_size) / (int)information[0]));
		float vz = -1 * information[4] + (information[10] + (globalIdx / tile_size)) / level_size;
		float ux = information[5];
		float uy = information[6];
		float uz = information[7];


		int lane = threadIdx.x % WARP_SIZE;
		int temp;
		while (globalIdx < num_elements)
		{
			// All threads in a block of 1024 take an element

			if (ux * vx + uy * vy + uz * vz >= 0) {
				temp = in[globalIdx] > 0 ? 1 : 0;
			}
			else {
				temp = 0;
			}

			// All warps in this block (32) compute the sum of all
			// threads in their warp
			for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
			{
				temp += __shfl_xor(temp, delta);
			}
			// Write all 32 of these partial sums to shared memory
			if (lane == 0)
			{
				buffer[threadIdx.x / WARP_SIZE] = temp;
			}
			__syncthreads();
			// Add the remaining 32 partial sums using a single warp
			if (threadIdx.x < WARP_SIZE)
			{
				temp = buffer[threadIdx.x];
				for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
				{
					temp += __shfl_xor(temp, delta);
				}
			}
			if (threadIdx.x == 0)
			{
				atomicAdd(sum, temp);
			}
			// Jump ahead 1024 * #SMs to the next region of numbers to sum
			globalIdx += blockDim.x * gridDim.x;
			__syncthreads();
		}
	}

}

extern "C"
__global__ void _touch_test(const unsigned char * subregion, unsigned int *cell, int *table, const float *information)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int tile_size = (int)information[0] * (int)information[0];
	int level_size = _pow_(2, (int)information[11]);
	//information : block_width, tablesize, pos_x,y,z , up_x,y,z , global_x,y,z
	if ((id / tile_size) % level_size == 0) {

		float vx = -1 * information[2] + (information[8] + ((int)(id % tile_size) % (int)information[0]));
		float vy = -1 * information[3] + (information[9] + ((int)(id % tile_size) / (int)information[0]));
		float vz = -1 * information[4] + (information[10] + (id / tile_size)) / level_size;
		float ux = information[5];
		float uy = information[6];
		float uz = information[7];

		if (ux * vx + uy * vy + uz * vz >= 0) {
			if (subregion[id] > 0) {
				if (cell[id] != 0) {
					table[cell[id]] = 1;
					cell[id] = 0;
				}
			}
		}
	}
}


extern "C"
__global__ void _intersect_test(unsigned int *cell, int *table, int *itable, const float *information)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int tile_size = (int)information[0] * (int)information[0];
	//information : block_width, tablesize, pos_x,y,z , up_x,y,z , global_x,y,z
	int level_size = _pow_(2, (int)information[11]);

	if ((id / tile_size) % level_size == 0) {
		float vx = -1 * information[2] + (information[8] + ((int)(id % tile_size) % (int)information[0]));
		float vy = -1 * information[3] + (information[9] + ((int)(id % tile_size) / (int)information[0]));
		float vz = -1 * information[4] + (information[10] + (id / tile_size)) / level_size;
		float ux = information[5];
		float uy = information[6];
		float uz = information[7];

		if (ux * vx + uy * vy + uz * vz >= 0) {
			if (cell[id] != 0) {
				if (table[cell[id]] == 1) {
					itable[cell[id]] = 1;
				}
			}
		}
	}

	
}

extern "C"
__global__ void _vector_add(int *table, int *itable)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	table[id] = table[id] + itable[id];
}



extern "C"
__global__ void _touch_test2(const unsigned char * subregion, unsigned int *cell, int *table, int baseBlockSize)
{

	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (subregion[id] > 0) {
		if (cell[id] != 0) {
			table[cell[id]]=2;
		}
	}
}
extern "C"
__global__ void _intersect_test2(const unsigned char * subregion, unsigned int *cell, int *table, int baseBlockSize)
{

	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (subregion[id] == 0) {
		if (table[cell[id]] == 2) {
			table[cell[id]] = 1;
		}
	}
}


extern "C"
__global__ void _cuda_parallel_type_decision(unsigned char *in, int num_elements, int *sum, const int *information)
{
	//Holds intermediates in shared memory reduction
	__syncthreads();
	__shared__ int buffer[WARP_SIZE];
	int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int tile_size = (int)information[0] * (int)information[0];
	//information : basic block size(512), block size, start pos_x,y,z

	int lane = threadIdx.x % WARP_SIZE;
	int temp;
	while (globalIdx < num_elements)
	{
		// All threads in a block of 1024 take an element
		temp = in[globalIdx] > 0 ? 1 : 0;

		// All warps in this block (32) compute the sum of all
		// threads in their warp
		for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
		{
			temp += __shfl_xor(temp, delta);
		}
		// Write all 32 of these partial sums to shared memory
		if (lane == 0)
		{
			buffer[threadIdx.x / WARP_SIZE] = temp;
		}
		__syncthreads();
		// Add the remaining 32 partial sums using a single warp
		if (threadIdx.x < WARP_SIZE)
		{
			temp = buffer[threadIdx.x];
			for (int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
			{
				temp += __shfl_xor(temp, delta);
			}
		}
		if (threadIdx.x == 0)
		{
			atomicAdd(sum, temp);
		}
		// Jump ahead 1024 * #SMs to the next region of numbers to sum
		globalIdx += blockDim.x * gridDim.x;
		__syncthreads();
	}

}


int cuda_parallel_sum(const unsigned char * a, const float *information) {
	// Get device properties to compute optimal launch bounds
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int N = (int)information[0] * (int)information[0] * (int)information[0];
	int num_SMs = prop.multiProcessorCount;
	// pad array with zeros to allow sum algorithm to work
	int batch_size = num_SMs * 1024;
	int padding = (batch_size - (N % batch_size)) % batch_size;
	// b is the new padded array
	unsigned char * b = new unsigned char[N + padding];
	memcpy(b, a, N * sizeof(unsigned char));
	memset(b + N, 0, padding * sizeof(unsigned char));

	// Copy array "b" to GPU 
	unsigned char *d_b;
	hipMalloc((void**)&d_b, (N + padding) * sizeof(unsigned char));
	hipMemcpy(d_b, b, (N + padding) * sizeof(unsigned char), hipMemcpyHostToDevice);

	float *d_information;
	hipMalloc((void**)&d_information, 12 * sizeof(float));
	hipMemcpy(d_information, information, 12 * sizeof(float), hipMemcpyHostToDevice);


	// Result
	int result = 0.0;
	int * d_result;
	hipMalloc((void**)&d_result, sizeof(int));
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

	// Call kernel to get sum
	_cuda_parallel_sum << < num_SMs, 1024 >> >(d_b, N + padding, d_result, d_information);
	// Read in results
	hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

	// Clean up
	hipFree(d_result);
	hipFree(d_b);
	hipFree(d_information);

	
	free(b);

	return result;
}

int* cuda_parallel_inclusion(const unsigned char * subregion, unsigned int *cell, const float *information) {
	//information : block_width, tablesize, pos_x,y,z , up_x,y,z , global_x,y,z
	int blockSize, gridSize;
	int N = (int)information[0] * (int)information[0] * (int)information[0];
	int t_size = (int)information[1];

	blockSize = 1024;
	gridSize = (int)ceil((float)N / blockSize);

	// Copy array to GPU 
	unsigned char *d_subregion;
	hipMalloc((void**)&d_subregion, N * sizeof(unsigned char));
	hipMemcpy(d_subregion, subregion, N * sizeof(unsigned char), hipMemcpyHostToDevice);

	unsigned int *d_cell;
	hipMalloc((void**)&d_cell, N * sizeof(unsigned int));
	hipMemcpy(d_cell, cell, N * sizeof(unsigned int), hipMemcpyHostToDevice);

	int *table = new int[t_size];
	memset(table, 0, sizeof(int) *t_size);
	int *d_table;
	hipMalloc((void**)&d_table, t_size * sizeof(int));
	hipMemcpy(d_table, table, t_size * sizeof(int), hipMemcpyHostToDevice);

	float *d_information;
	hipMalloc((void**)&d_information, 12 * sizeof(float));
	hipMemcpy(d_information, information, 12 * sizeof(float), hipMemcpyHostToDevice);


	// Call kernel to get sum
	_touch_test <<< gridSize, blockSize >>>(d_subregion, d_cell, d_table, d_information);
	
	
	int *table_i = new int[t_size];
	memset(table_i, 0, sizeof(int) *t_size);
	int *d_itable;
	hipMalloc((void**)&d_itable, t_size * sizeof(int));
	hipMemcpy(d_itable, table_i, t_size * sizeof(int), hipMemcpyHostToDevice);

	_intersect_test <<< gridSize, blockSize >>>(d_cell, d_table, d_itable, d_information);


	blockSize = 1024;
	gridSize = (int)ceil((float)t_size / blockSize);

	_vector_add << < gridSize, blockSize >> > (d_table, d_itable);

	
	hipMemcpy(table, d_table, t_size * sizeof(int), hipMemcpyDeviceToHost);
	
	// Clean up
	hipFree(d_subregion);
	hipFree(d_cell);
	hipFree(d_table);
	hipFree(d_itable);
	hipFree(d_information);

	return table;
}

int* cuda_parallel_cell_type_decision(const unsigned char * subregion, unsigned int *cell, int baseBlockSize,int cell_number) {
	int blockSize, gridSize;
	int N = baseBlockSize*baseBlockSize*baseBlockSize;
	int t_size = cell_number;

	blockSize = 1024;
	gridSize = (int)ceil((float)N / blockSize);

	// Copy array to GPU 
	unsigned char *d_subregion;
	hipMalloc((void**)&d_subregion, N * sizeof(unsigned char));
	hipMemcpy(d_subregion, subregion, N * sizeof(unsigned char), hipMemcpyHostToDevice);

	unsigned int *d_cell;
	hipMalloc((void**)&d_cell, N * sizeof(unsigned int));
	hipMemcpy(d_cell, cell, N * sizeof(unsigned int), hipMemcpyHostToDevice);

	int *table = new int[t_size];
	memset(table, 0, sizeof(int) *t_size);
	int *d_table;
	hipMalloc((void**)&d_table, t_size * sizeof(int));
	hipMemcpy(d_table, table, t_size * sizeof(int), hipMemcpyHostToDevice);


	// Call kernel to get sum
	_touch_test2 <<< gridSize, blockSize >>>(d_subregion, d_cell, d_table, baseBlockSize);
	_intersect_test2 <<< gridSize, blockSize >>>(d_subregion, d_cell, d_table, baseBlockSize);

	
	
	hipMemcpy(table, d_table, t_size * sizeof(int), hipMemcpyDeviceToHost);
	
	// Clean up
	hipFree(d_subregion);
	hipFree(d_cell);
	hipFree(d_table);

	return table;
}


int cuda_parallel_type_decision(const unsigned char * subregion, const int *information) {
	// Get device properties to compute optimal launch bounds
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	//information: 0:basic block size, 1:block size, 2:start x, 3:start y, 4: start z
	int N = information[0]*information[0]*information[0];
	int num_SMs = prop.multiProcessorCount;
	// pad array with zeros to allow sum algorithm to work
	int batch_size = num_SMs * 1024;
	int padding = (batch_size - (N % batch_size)) % batch_size;
	// b is the new padded array
	unsigned char * b = new unsigned char[N + padding];
	memcpy(b, subregion, N * sizeof(unsigned char));
	memset(b + N, 0, padding * sizeof(unsigned char));

	// Copy array "b" to GPU 
	unsigned char *d_b;
	hipMalloc((void**)&d_b, (N + padding) * sizeof(unsigned char));
	hipMemcpy(d_b, b, (N + padding) * sizeof(unsigned char), hipMemcpyHostToDevice);

	int *d_information;
	hipMalloc((void**)&d_information, 5 * sizeof(int));
	hipMemcpy(d_information, information, 5 * sizeof(int), hipMemcpyHostToDevice);


	// Result
	int result = 0.0;
	int * d_result;
	hipMalloc((void**)&d_result, sizeof(int));
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

	// Call kernel to get sum
	_cuda_parallel_type_decision << < num_SMs, 1024 >> >(d_b, N + padding, d_result, d_information);
	// Read in results
	hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

	// Clean up
	hipFree(d_result);
	hipFree(d_b);
	free(b);

	if(result==0)return 0;
	else if(result==information[1]*information[1]*information[1])return 2;
	else return 1;
}
